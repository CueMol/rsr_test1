#include "hip/hip_runtime.h"
// -*-Mode: C++;-*-

#include <stdio.h>
#include <vector>

#include "cudacode.h"
 
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}



#include "cuda_map_kern.hpp"

void cudaMap_fdf1(const std::vector<float> &crds,
		 CudaMapData *pDat,
		 float *val, std::vector<float> &grad)
{
  const int ncrds = crds.size();
  const int natom = ncrds/3;
  const int ntoth = pDat->nblk * pDat->nthr;

  // Coordinates array (crds)
  if (pDat->pd_crds==NULL)
    hipMalloc((void**)&pDat->pd_crds, ncrds*sizeof(float));
  hipMemcpy( pDat->pd_crds, &crds[0], ncrds*sizeof(float), hipMemcpyHostToDevice);
  //printf("CUDA crds (%d*%d) = %p\n", ncrds, sizeof(float), pDat->pd_crds);

  // Weights array (wgts)
  if (pDat->pd_wgts==NULL) {
    hipMalloc((void**)&pDat->pd_wgts, ntoth*sizeof(float));
    hipMemcpy( pDat->pd_wgts, &pDat->wgts[0], ntoth*sizeof(float), hipMemcpyHostToDevice);
    //for (int i=0; i<ntoth; ++i) {
    //printf("wgts: %d %f\n", i, pDat->wgts[i]);
    //}
    hipMemcpyToSymbol(HIP_SYMBOL(g_fracMat1), &pDat->fracMat[0], 9*sizeof(float));
  }

  // Density map (3D texture)
  hipChannelFormatDesc cdesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  if (pDat->pd_map==NULL) {
    hipExtent ext = make_hipExtent(pDat->ncol,pDat->nrow,pDat->nsec);
    gpuErrChk(hipMalloc3DArray((hipArray_t *)&pDat->pd_map, &cdesc, ext));
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(&pDat->p_map[0], pDat->ncol*sizeof(float), pDat->ncol,pDat->nrow);
    copyParams.dstArray = (hipArray_t)pDat->pd_map;
    copyParams.extent   = ext;
    copyParams.kind     = hipMemcpyHostToDevice;
    gpuErrChk(hipMemcpy3D(&copyParams));
    gpuErrChk(hipBindTextureToArray(texRef1, (hipArray_t)pDat->pd_map, cdesc)); 
  }

  // result: grad vec
  if (pDat->pd_grad==NULL) {
    //hipMalloc((void**)&pDat->pd_grad, ncrds*sizeof(float));
    hipMalloc((void**)&pDat->pd_grad, ntoth*3*sizeof(float));
    hipMalloc((void**)&pDat->pd_eatm, pDat->nblk*sizeof(float));
  }
  
  const int nshmem = pDat->nthr * sizeof(float);
  MapGradKern1<<<pDat->nblk, pDat->nthr, nshmem>>>
    (pDat->pd_crds, pDat->pd_wgts,
     make_float4(pDat->na, pDat->nb, pDat->nc, 1.0),
     make_float4(pDat->stcol, pDat->strow, pDat->stsec, 1.0),
     pDat->pd_grad, pDat->pd_eatm);

  //printf("kern exec OK\n");

  hipDeviceSynchronize();

  //printf("kern synch OK\n");

  hipMemcpy( &grad[0], pDat->pd_grad, natom*3*sizeof(float), hipMemcpyDeviceToHost);

#if 0
  std::vector<float> gradtmp(natom*4);
  hipMemcpy( &gradtmp[0], pDat->pd_grad, natom*4*sizeof(float), hipMemcpyDeviceToHost);

  //*val = 0.0f;
  for (int i=0;i<natom; ++i) {
    /*union {
      float f;
      unsigned int ui;
    } u;
    u.f = grad[i*4+3];
    printf("  eng.x %d  %.16e [%x]\n", i, u.f, u.ui);*/
    grad[i*3+0] = gradtmp[i*4+0];
    grad[i*3+1] = gradtmp[i*4+1];
    grad[i*3+2] = gradtmp[i*4+2];
    //*val += gradtmp[i*4+3];
  }
#endif

  hipMemcpy( &pDat->eatom[0], pDat->pd_eatm, pDat->nblk*sizeof(float), hipMemcpyDeviceToHost);
  *val = 0.0f;
  for (int i=0; i<pDat->nblk; ++i)
    *val += pDat->eatom[i];


  //hipUnbindTexture(texRef1); 

  //printf("Results copy OK\n");
}


#include "cuda_map_kern2.hpp"

void cudaMap_fdf2(const std::vector<float> &crds,
		  CudaMapData *pDat,
		  float *val, std::vector<float> &grad)
{
  const int ncrds = crds.size();
  const int natom = ncrds/3;

  // Coordinates array (crds)
  if (pDat->pd_crds==NULL)
    hipMalloc((void**)&pDat->pd_crds, ncrds*sizeof(float));
  hipMemcpy( pDat->pd_crds, &crds[0], ncrds*sizeof(float), hipMemcpyHostToDevice);

  // Weights array (wgts)
  if (pDat->pd_wgts==NULL) {
    int nwgt = pDat->wgts.size();
    hipMalloc((void**)&pDat->pd_wgts, nwgt*sizeof(float));
    hipMemcpy( pDat->pd_wgts, &pDat->wgts[0], nwgt*sizeof(float), hipMemcpyHostToDevice);
    //for (int i=0; i<natom; ++i) {
    //printf("wgts: %d %f\n", i, pDat->wgts[i]);
    //}

    hipMemcpyToSymbol(HIP_SYMBOL(g_fracMat2), &pDat->fracMat[0], 9*sizeof(float));

    float gcoef[] = {
      -0.5,  0.0, 0, 0, 0,
      0.0,  -1.5, 1, 1, 0,
      -1.5,  0.0, 1, 0, 1,
      0.0,  -0.5, 0, 0, 0,
    };
    hipMemcpyToSymbol(HIP_SYMBOL(g_coefMat2), gcoef, 20*sizeof(float));
  }

  // Density map (3D texture)
  hipChannelFormatDesc cdesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  if (pDat->pd_map==NULL) {
    hipExtent ext = make_hipExtent(pDat->ncol,pDat->nrow,pDat->nsec);
    gpuErrChk(hipMalloc3DArray((hipArray_t *)&pDat->pd_map, &cdesc, ext));
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(&pDat->p_map[0], pDat->ncol*sizeof(float), pDat->ncol,pDat->nrow);
    copyParams.dstArray = (hipArray_t)pDat->pd_map;
    copyParams.extent   = ext;
    copyParams.kind     = hipMemcpyHostToDevice;
    gpuErrChk(hipMemcpy3D(&copyParams));
    gpuErrChk(hipBindTextureToArray(texRef2, (hipArray_t)pDat->pd_map, cdesc)); 
  }

  // result: grad vec/Eatom
  if (pDat->pd_grad==NULL) {
    hipMalloc((void**)&pDat->pd_grad, ncrds*sizeof(float));
    hipMalloc((void**)&pDat->pd_eatm, pDat->nblk*sizeof(float));
  }
  
  const int nshmem = pDat->nthr * sizeof(float);
  MapGradKern2<<<pDat->nblk, pDat->nthr, nshmem>>>
    (pDat->pd_crds, pDat->pd_wgts,
     make_float4(pDat->na, pDat->nb, pDat->nc, 1.0),
     make_float4(pDat->stcol, pDat->strow, pDat->stsec, 1.0),
     pDat->pd_grad, pDat->pd_eatm);

  //printf("kern exec OK\n");

  hipDeviceSynchronize();

  //printf("kern synch OK\n");

  hipMemcpy( &grad[0], pDat->pd_grad, ncrds*sizeof(float), hipMemcpyDeviceToHost);

  hipMemcpy( &pDat->eatom[0], pDat->pd_eatm, pDat->nblk*sizeof(float), hipMemcpyDeviceToHost);

  *val = 0.0f;

  for (int i=0; i<pDat->eatom.size(); ++i) {
    /*
    int x = i%64;
    int ii = x/16;
    int jj = (x%16)/4;
    int kk = x%4;
    printf("%d %d %d %e\n", ii, jj, kk, pDat->eatom[i]);*/
    *val += pDat->eatom[i];
  }

  //hipUnbindTexture(texRef2); 

  //printf("Results copy OK\n");
}
